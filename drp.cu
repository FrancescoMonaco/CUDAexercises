#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

class DiscretizedRandomProjections{
  public:
    DiscretizedRandomProjections(int dim, int r, int K, int L):
    dim(dim), r(r), K(K), L(L){
      sqrtL = int(sqrt(L));
      K_half = int(K/2);

      // Initialize a_l and a_r as 3d cuda int arrays of size sqrtL, K_half, dim
      hipMalloc3DArray(&a_l, make_hipExtent(sqrtL, K_half, dim));
      hipMalloc3DArray(&a_r, make_hipExtent(sqrtL, K_half, dim));

      // Initialize b_l and b_r as 2d cuda int arrays of size sqrtL, K_half
      hipMalloc3DArray(&b_l, make_hipExtent(sqrtL, K_half, 0));
      hipMalloc3DArray(&b_r, make_hipExtent(sqrtL, K_half, 0));

      // Fill with the random values using the kernel function
      int blockSize = 256;
      int numBlocks = (dim + blockSize - 1) / blockSize;
      init_random<<<numBlocks, blockSize>>>();   
      
      // Wait for GPU to finish before accessing on host
      hipDeviceSynchronize();

    }

    ~DiscretizedRandomProjections(){
      hipFreeArray(a_l);
      hipFreeArray(a_r);
      hipFreeArray(b_l);
      hipFreeArray(b_r);
    }

    __global__
    void init_random(DiscretizedRandomProjections rp);

  __global__
   auto compute_hash(const int[] sub, const DiscretizedRandomProjections rp);

  private:
    int dim;
    int r;
    int K;
    int L;
    int sqrtL;
    int K_half;
    float* a_l, b_l, a_r, b_r;




}



void main(int argc, char** argv){

  return;
}

DiscretizedRandomProjections::init_random(DiscretizedRandomProjections rp){
  // Find the indices for all the arrays
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  int stride = blockDim.x * gridDim.x;

  // Fill the arrays with random values
  for(int i = x; i < rp.sqrtL; i += stride){
    for(int j = y; j < rp.K_half; j += stride){
      for(int k = z; k < rp.dim; k += stride){
        rp.a_l[i][j][k] = rand();
        rp.a_r[i][j][k] = rand();
      }

      rp.b_l[i][j] = rand();
      rp.b_r[i][j] = rand();
    }
  }

  return;
}